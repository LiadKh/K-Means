#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#define THREAD_IN_BLOCK 1024

__global__ void inicDTKernel(float *inicedPoints, const float *points, float dt, int dim, int numberOfPoints)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < numberOfPoints)
	{
		for (int i = 0; i < dim; i++) {
			inicedPoints[index * dim + i] = points[index * dim * 2 + i] + dt * points[index * dim * 2 + i + dim];
		
		}
	}
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t inicDTWithCuda(float* points, int numberOfPoints, int dim, float dt, float** inicedPoints)
{
	float *dev_points = 0;
	float *dev_iniced_points = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_points, numberOfPoints *dim * 2 * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_iniced_points, numberOfPoints *dim * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_points, points, numberOfPoints *dim * 2 * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	int numberOfBlock = numberOfPoints / THREAD_IN_BLOCK + 1;
	if (numberOfPoints % THREAD_IN_BLOCK != 0)
		numberOfBlock++;
	// Launch a kernel on the GPU with one thread for each element.
	inicDTKernel << <numberOfBlock, THREAD_IN_BLOCK >> > (dev_iniced_points, dev_points, dt, dim, numberOfPoints);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(*inicedPoints, dev_iniced_points, numberOfPoints *dim * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_points);
	hipFree(dev_iniced_points);

	return cudaStatus;
}
