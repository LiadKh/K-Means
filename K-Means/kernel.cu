#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "Const.h"

#define THREAD_IN_BLOCK 1000
#define MAX_CLUSTERS 200
#define ONE_THREAD_WORK 5

__global__ void incKernel(point_t *incPoints, const point_t *points, double dT, int numberOfPoints)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < numberOfPoints)
	{
		incPoints[index].x = points[index].x + dT*points[index].vx;
		incPoints[index].y = points[index].y + dT*points[index].vy;
		incPoints[index].z = points[index].z + dT*points[index].vz;
	}
}

__device__ double distance2Points(point_t *p1, point_t *p2)
{//Find distance between two points
	return sqrt(pow(p1->x - p2->x, 2) + pow(p1->y - p2->y, 2) + pow(p1->z - p2->z, 2));
}

__global__ void setCloseClusterKernel(point_t *points, int numberOfPoints, point_t *clusters, int numberOfClusters)
{
	int index = (blockIdx.x * blockDim.x + threadIdx.x)*ONE_THREAD_WORK;
	double temp, distance;
	__shared__ point_t sharedClusters[MAX_CLUSTERS];
	if (threadIdx.x < numberOfClusters)
		sharedClusters[threadIdx.x] = clusters[threadIdx.x];
	__syncthreads();
	for (int i = 0; i < ONE_THREAD_WORK && index < numberOfPoints; i++, index++)
	{
		for (int j = 0; j < numberOfClusters; j++)
		{
			temp = distance2Points(&(points[index]), &(sharedClusters[j]));
			if (j == 0 || temp < distance)
			{//Set close cluster id
				points[index].cluster = j;
				distance = temp;
			}
		}
	}
}

// Helper function for using CUDA to inc point with dt speed in parallel.
hipError_t incPointsWithCuda(point_t* points, int numberOfPoints, double dT, point_t* incPoints)
{
	point_t *dev_points = 0;
	point_t *dev_iniced_points = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_points, numberOfPoints * sizeof(point_t));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_iniced_points, numberOfPoints * sizeof(point_t));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_points, points, numberOfPoints * sizeof(point_t), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	incKernel << <numberOfPoints / THREAD_IN_BLOCK + 1, THREAD_IN_BLOCK >> > (dev_iniced_points, dev_points, dT, numberOfPoints);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "incKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(incPoints, dev_iniced_points, numberOfPoints * sizeof(point_t), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_points);
	hipFree(dev_iniced_points);

	return cudaStatus;
}

// Helper function for using CUDA to set the close cluster to each point in parallel..
hipError_t setCloseClusterWithCuda(point_t* points, int numberOfPoints, point_t* clusters, int numberOfClusters)
{
	point_t *dev_points = 0;
	point_t *dev_clusters = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_points, numberOfPoints * sizeof(point_t));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_clusters, numberOfClusters * sizeof(point_t));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_points, points, numberOfPoints * sizeof(point_t), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_clusters, clusters, numberOfClusters * sizeof(point_t), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	//int numberOfThread = THREAD_IN_BLOCK / ONE_THREAD_WORK;
	setCloseClusterKernel << <numberOfPoints / (THREAD_IN_BLOCK / ONE_THREAD_WORK) + 1, THREAD_IN_BLOCK >> > (dev_points, numberOfPoints, dev_clusters, numberOfClusters);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "setCloseClusterKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(points, dev_points, numberOfPoints * sizeof(point_t), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_points);
	hipFree(dev_clusters);

	return cudaStatus;
}