#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "Const.h"

#define THREAD_IN_BLOCK 1024

__global__ void incDTKernel(point_t *inicedPoints, const point_t *points, float dT, int numberOfPoints)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < numberOfPoints)
	{
		inicedPoints[index].x = points[index].x + dT*points[index].vx;
		inicedPoints[index].y = points[index].y + dT*points[index].vy;
		inicedPoints[index].z = points[index].z + dT*points[index].vz;
	}
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t incDTWithCuda(point_t* points, int numberOfPoints, float dT, point_t* inicedPoints)
{
	point_t *dev_points = 0;
	point_t *dev_iniced_points = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_points, numberOfPoints * sizeof(point_t));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_iniced_points, numberOfPoints * sizeof(point_t));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_points, points, numberOfPoints * sizeof(point_t), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	int numberOfBlock = numberOfPoints / THREAD_IN_BLOCK + 1;
	if (numberOfPoints % THREAD_IN_BLOCK != 0)
		numberOfBlock++;
	// Launch a kernel on the GPU with one thread for each element.
	incDTKernel << <numberOfBlock, THREAD_IN_BLOCK >> > (dev_iniced_points, dev_points, dT, numberOfPoints);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(inicedPoints, dev_iniced_points, numberOfPoints * sizeof(point_t), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_points);
	hipFree(dev_iniced_points);

	return cudaStatus;
}
